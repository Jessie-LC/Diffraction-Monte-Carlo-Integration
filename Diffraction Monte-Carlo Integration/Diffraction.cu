#include "hip/hip_runtime.h"
#include "Diffraction.cuh"

using namespace glm;

__device__ struct RNG_State {
    uint64_t x, w1, s;
};

__device__ unsigned msws(RNG_State& rng) {
    rng.x *= rng.x;
    rng.x += (rng.w1 += rng.s);
    return unsigned(rng.x = (rng.x >> 32u) | (rng.x << 32u));

}
__device__ void init_msws(uint64_t seed, RNG_State& rng) {
    rng.x = 0u; rng.w1 = 0u;
    rng.s = (((uint64_t(1890726812u) << 32u) | seed) << 1u) | uint64_t(1u);

    msws(rng); msws(rng);
}

#define RandNext(rng) msws(rng)
#define RandNext2(rng) uvec2(msws(rng), msws(rng))
#define RandNext3(rng) uvec3(RandNext2(rng), msws(rng))
#define RandNext4(rng) uvec4(RandNext3(rng), msws(rng))

#define RandNextF(rng) (float(RandNext(rng) & 0x00ffffffu) / float(0x00ffffff))
#define RandNext2F(rng) (vec2(RandNext2(rng) & 0x00ffffffu) / float(0x00ffffff))
#define RandNext3F(rng) (vec3(RandNext3(rng) & 0x00ffffffu) / float(0x00ffffff))
#define RandNext4F(rng) (vec4(RandNext4(rng) & 0x00ffffffu) / float(0x00ffffff))

__constant__ float pi = 3.14159;
__constant__ float tau = 6.28318;
__constant__ float phi = 1.61803399;

__device__ mat2 Rotate(float a) {
    float s = sin(a);
    float c = cos(a);
    return mat2(c, -s, s, c);
}

__device__ vec2 BokehShape(RNG_State& rng, int bladeCount, float radius) {
    const int blades = bladeCount;

    vec2 uv = RandNext2F(rng);

    vec2 axis;
    const float angle = radians(360.0f) / float(blades);

    uv.x *= float(blades);
    float blade = floor(uv.x);
    uv.x = fract(uv.x);

    mat2 rot = Rotate(blade * angle);

    axis = rot * vec2(cos(angle / 2.0), sin(angle / 2.0) * (uv.x * 2.0 - 1.0));
    axis *= 1.0 - pow(1.0 - sqrt(uv.y), 1.0);

    return axis * radius;
}

__device__ vec2 SampleCircle(RNG_State& rngState, float radius) {
    vec2 rng = RandNext2F(rngState);
    float r = radius * sqrt(rng.x);
    float t = 2.0 * pi * rng.y;

    return r * vec2(cos(t), sin(t));
}

__device__ float absSquared(thrust::complex<float> value) {
    return value.real() * value.real() + value.imag() * value.imag();
}

__global__ void DiffractionIntegral(float* diff, int wavelengthIndex, DiffractionSettings settings) {
    int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int x = globalThreadIndex % settings.size;
    int y = globalThreadIndex / settings.size;

    unsigned s = unsigned(x * settings.size + y) * 720720u + unsigned(wavelengthIndex);

    RNG_State rng;
    init_msws(uint32_t(s), rng);

    //Everything is in micrometers
    float scale = settings.scale;
    float radius = settings.radius;
    float dist = settings.dist;

    //ensure scaleWeight + radiusWeight + distanceWeight == 1.0.
    float average = dot(
        vec3(scale, radius, dist),
        vec3(0.1f, 0.3f, 0.6f)
    );
    float deviation = sqrt(
        dot(
            pow(vec3(scale, radius, dist) - average, vec3(2.0f)),
            vec3(0.1f, 0.3f, 0.6f)
        )
    );

    float wavelength = ((441.0f * (float(wavelengthIndex) / (settings.wavelengthCount - 1))) + 390.0f) * 1e-3f;
    float k = 2.0f * pi / wavelength;

    float angle = pi / float(settings.bladeCount);
    float sinAngle = sin(angle);
    float cosAngle = cos(angle);
    float blades = float(settings.bladeCount);

    int steps = int(scale * pow(radius, 2.0f) * pow(dist, 2.0f) * settings.quality);
    thrust::complex<float> integral = thrust::complex<float>(0.0f, 0.0f);
    for (int i = 0; i < steps; ++i) {
        vec2 uv = scale * ((vec2(x, y) / vec2(settings.size, settings.size)) - 0.5f);
        vec2 rngUV = BokehShape(rng, blades, radius);

        float r = length(vec3(uv - rngUV, dist));
        float rk = r * k;

        thrust::complex<float> term = thrust::complex<float>(cos(rk), sin(rk)) * (dist / (r * r));

        if (isnan(term.real())) {
            term = thrust::complex<float>(0.0f, 0.0f);
        }
        if (isnan(term.imag())) {
            term = thrust::complex<float>(0.0f, 0.0f);
        }
        if (isinf(term.real())) {
            term = thrust::complex<float>(1.0f, 0.0f);
        }
        if (isinf(term.imag())) {
            term = thrust::complex<float>(0.0f, 1.0f);
        }

        integral += term;
    }
    integral /= steps;

    integral *= thrust::complex<float>(1.0f, -1.0f / wavelength);

    diff[x + y * settings.size] = absSquared(integral);
}