#include "hip/hip_runtime.h"
#include "Diffraction.cuh"

using namespace glm;

__device__ struct RNG_State {
    uint64_t x, w1, s;
};

__device__ unsigned msws(RNG_State& rng) {
    rng.x *= rng.x;
    rng.x += (rng.w1 += rng.s);
    return unsigned(rng.x = (rng.x >> 32u) | (rng.x << 32u));

}
__device__ void init_msws(uint64_t seed, RNG_State& rng) {
    rng.x = 0u; rng.w1 = 0u;
    rng.s = (((uint64_t(1890726812u) << 32u) | seed) << 1u) | uint64_t(1u);

    msws(rng); msws(rng);
}

#define RandNext(rng) msws(rng)
#define RandNext2(rng) uvec2(msws(rng), msws(rng))
#define RandNext3(rng) uvec3(RandNext2(rng), msws(rng))
#define RandNext4(rng) uvec4(RandNext3(rng), msws(rng))

#define RandNextF(rng) (float(RandNext(rng) & 0x00ffffffu) / float(0x00ffffff))
#define RandNext2F(rng) (vec2(RandNext2(rng) & 0x00ffffffu) / float(0x00ffffff))
#define RandNext3F(rng) (vec3(RandNext3(rng) & 0x00ffffffu) / float(0x00ffffff))
#define RandNext4F(rng) (vec4(RandNext4(rng) & 0x00ffffffu) / float(0x00ffffff))

__constant__ float pi = 3.14159;
__constant__ float tau = 6.28318;
__constant__ float phi = 1.61803399;

__device__ mat2 Rotate(float a) {
    vec2 m;
    m.x = sin(a);
    m.y = cos(a);
    return mat2(m.y, -m.x, m.x, m.y);
}

__device__ vec2 BokehShape(RNG_State& rng) {
    const int blades = 3;

    vec2 uv = RandNext2F(rng);

    vec2 axis;
    const float angle = radians(360.0f) / float(blades);

    uv.x *= float(blades);
    float blade = floor(uv.x);
    uv.x = fract(uv.x);

    mat2 rot = Rotate(blade * angle);

    axis = rot * vec2(cos(angle / 2.0), sin(angle / 2.0) * (uv.x * 2.0 - 1.0));
    axis *= 1.0 - pow(1.0 - sqrt(uv.y), 1.0);

    return axis;
}

__global__ void DiffractionIntegral(thrust::complex<float>* diff, int wavelengthIndex, DiffractionSettings settings) {
    int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int x = globalThreadIndex % settings.size;
    int y = globalThreadIndex / settings.size;

    unsigned s = unsigned(x * settings.size + y) * 720720u;

    RNG_State rng;
    init_msws(uint32_t(s), rng);

    //Everything is in micrometers
    float scale = settings.scale;
    float radius = settings.radius;
    float dist = settings.dist;

    float wavelength = ((441.0f * (float(wavelengthIndex) / (wavelengthCount - 1))) + 390.0f) * 1e-3f;

    int steps = int((pow(radius, 2.0f) * pow(dist, 2.0f)) * settings.quality);
    thrust::complex<float> integral = thrust::complex<float>(0.0f, 0.0f);
    for (int i = 0; i < steps; ++i) {

        vec2 uv = scale * ((vec2(x, y) / vec2(settings.size, settings.size)) - 0.5f);
        vec2 rngUV = BokehShape(rng) * radius;

        float k = 2.0f * pi / wavelength;
        float r = length(vec3(uv, dist) - vec3(rngUV, 0.0f));

        thrust::complex<float> term = (thrust::exp(thrust::complex<float>(0.0f, 1.0f) * (r * k)) / r) * (dist / r);

        if (isnan(term.real())) {
            term = thrust::complex<float>(0.0f, 0.0f);
        }
        if (isnan(term.imag())) {
            term = thrust::complex<float>(0.0f, 0.0f);
        }
        if (isinf(term.real())) {
            term = thrust::complex<float>(1.0f, 0.0f);
        }
        if (isinf(term.imag())) {
            term = thrust::complex<float>(0.0f, 1.0f);
        }

        integral += term * (1.0f / steps);
    }

    integral = (thrust::complex<float>(1.0f, 0.0f) / (thrust::complex<float>(0.0f, 1.0f) * wavelength)) * integral;

    diff[x + y * settings.size] = integral;
}